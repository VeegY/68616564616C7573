#include "hip/hip_runtime.h"
﻿// #################################################################################################
//			Studienprojekt Modellbildung & Simulation - 2015/16
// #################################################################################################
// 					zufallstest.cpp
// ------------------------------------Doxygen-Dokumentation----------------------------------------
///  \file randtest.cpp
///  \brief
///  Testet spmv, gpudefect, (spmv2) und vergleicht die ergebnisse mit den Implementationen aus DIA.hpp 
///  
//#################################################################################################

#include"gpumatoperations.hpp"
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <time.h>   

using namespace std;
// testet  spmv, gpudefect für eine Matrix mit vorgegebener Bandstruktur, vorgegebenem Datentyp und Einträgen zwischen -100 und 100
template<typename type>
void generatetest(int dim, int ndiags, Vector<int>& offset)
{
    Vector<type> x(dim);
    Vector<type> b(dim);
    Vector<type> res(dim);
    Vector<type> data (dim*ndiags);
    srand (time(NULL));
	//setze x Daten
    for (int i=0; i< dim; ++i){
        x[i]=(static_cast <type> ((rand()) / static_cast <type> (RAND_MAX))*200 - 100);
    }
    //setze Matrix Daten
    for (int i=0; i< ndiags; ++i){
        if(offset[i]<=0){
            for (int j=-offset[i]; j< dim; ++j){
                data[i*dim+j]=((static_cast <type> (rand()) / static_cast <type> (RAND_MAX))*200 - 100);
            }
        }else
        {
            for (int j=0; j< dim-offset[i]; ++j){
                data[i*dim+j]=((static_cast <type> (rand()) / static_cast <type> (RAND_MAX))*200 - 100);
            }
        }
    }

    DIA<type> mat (dim, ndiags, data, offset);
    cout<<"Matrix erstellt"<<endl;
    //cout<<mat.checkIntact()<<endl;

    spmv(b, mat, x);
    defect(res, mat, b, x);
    cout<<"norm gpu ergebnis: "<<norm(b)<<endl;
    cout<<"norm defekt: "<<norm(res)<<endl;
    gpudefect(res, mat, b, x);
    cout<<"norm gpudefekt: "<<norm(res)<<endl;
	

    matvec(b, mat, x);
	gpudefect(res, mat, b, x);
	cout<<"norm cpu ergebnis: "<<norm(b)<<endl;
    cout<<"norm gpudefekt: "<<norm(res)<<endl;
    defect(res, mat, b, x);
    cout<<"norm defekt: "<<norm(res)<<endl;

}
int main (){
    int dim (1000000);
    int ndiags(7);
    Vector<int> offset(ndiags);
    offset[0]=-10000;
    offset[1]=-100;
    offset[2]=-1;
    offset[3]=0;
    offset[4]=1;
    offset[5]=100;
    offset[6]=10000;
    
    generatetest<double>(dim, ndiags, offset);
}


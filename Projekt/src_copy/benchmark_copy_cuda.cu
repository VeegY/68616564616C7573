#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <string>
#include "include/timer.hpp"
#define RESET "\e[0m"
#define BLUE "\e[34;1m"
#define CYAN "\e[36;1m"
#define GREY "\e[30;1m"
#define MAGENTA "\e[35;1m"

template <typename Scalar>
void cleanup(Scalar *pointer, int method);

//KERNEL!!!
template<typename type>
__global__ void gpu_copy(type *one, type *two, int dim_local)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < dim_local)
    {
        two[idx] = one[idx];
    }
    
}

//CHANGE!!!!
template<typename type>
void performance(float time_ku, float time_ou, float time_kz, float time_oz, int runs, type schalter, int meth, int ver_first, int ver_second, int mem_option, int dim_local)
{
    using std::string;
    string first, second, method;
    string memop = "";
    if (mem_option == 0) { memop = "(Unified Memory)"; }
    else { memop = "(Zero Copy)"; }
    
    if (meth == 0)
    {
        method = "Unified Memory vs Zero Copy";
        first = "Unified Memory";
        second = "Zero Copy";
    }
    if (meth == 1)
    {
        method = "Kernel vs Kernel";
        first = "Kernel Version" + memop + ": " ;
        first += ver_first;
        second = "Kernel Version" + memop + ": ";
        second += ver_second;

    }
    if (meth == 2)
    {
        method = "Kernel vs CPU";
        first = "Kernel Version" + memop + ": ";
        first += ver_first;
        second = "CPU";
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);

    //CHANGE BELOW HERE!!!
    //===#ELEMENTS IN THE VECTOR===================================//
    unsigned long long int elements = dim_local;

    //==='DISK STORAGE~============================================//
    unsigned long long int storage = sizeof(type)*dim_local;
    
    //===#FLOP=====================================================//
    int num_threads = 1024;
    if (dim_local<1024)
    {
        int n = dim_local-1;
        n |= n >> 1;
        n |= n >> 2;
        n |= n >> 4;
        n |= n >> 8;
        n |= n >> 16;
        n |= n >> 16;
        n |= n >> 16;
        num_threads = n + 1;
    }
    int num_blocks = ceil((double)dim_local / 1024);
    
    unsigned long long int flop = elements;         //MULT INTO SHARED MEMORY
    for(int i=2;i<num_threads;i=i*2)
    {
        elements += i*num_blocks;                   //REDUCE KERNEL
    }
    elements += num_blocks;                         //PLACEHOLDER REDUCE

    //==#BYTES=====================================================//           
    int bytes = (elements+1)*sizeof(type);


    printf(GREY "===============================================\n");
    printf(MAGENTA "                PERFORMANCE\n");
    printf("           %s\n", method.c_str());
    printf("        DIM = %i ~~ %i Iterations\n", dim_local, runs);
    printf("            %.2fGB/2GB DRAM used\n", storage / 1.0e9);
    printf(GREY "===============================================\n");
    printf("-----------------------------------------------\n");
    printf(CYAN "                    %s\n", first.c_str());
    printf(GREY "-----------------------------------------------\n");
    printf(CYAN "Kernel Runtime:\t\t\t%f(ms)\n",time_ku);
    printf("Overall Runtime:\t\t%f(ms)\n",time_ou*1.0e3);
    printf("Bandwith(th. Peak):\t\t%.2f(14.9)(GB/s)\n", bytes / (time_ku*1.0e6));
    printf("Flops(th. Peak):\t\t%.6f(326)(GFLOPS/s)\n", flop  / (time_ku*1.0e6));
    printf(GREY "-----------------------------------------------\n");
    printf("-----------------------------------------------\n");
    printf(BLUE "                     %s\n", second.c_str());
    printf(GREY "-----------------------------------------------\n");
    printf(BLUE "Kernel Runtime:\t\t\t%f(ms)\n",time_kz);
    printf("Overall Runtime:\t\t%f(ms)\n",time_oz*1.0e3);
    printf("Bandwith(th. Peak):\t\t%.2f(14.9)(GB/s)\n", bytes / (time_kz*1.0e6));
    printf("Flops(th. Peak):\t\t%.6f(326)(GFLOPS/s)\n", flop  / (time_kz*1.0e6));
    printf(GREY "-----------------------------------------------\n" RESET);



    
}
template void performance<int>(float time_ku, float time_ou, float time_kz, float time_oz, int runs, int schalter, int meth, int ver_first, int ver_second, int mem_option, int dim_local);
template void performance<float>(float time_ku, float time_ou, float time_kz, float time_oz, int runs, float schalter, int meth, int ver_first, int ver_second, int mem_option, int dim_local);
template void performance<double>(float time_ku, float time_ou, float time_kz, float time_oz, int runs, double schalter, int meth, int ver_first, int ver_second, int mem_option, int dim_local);


//PROPERTIES OF TEGRA K1
void print_p()
{

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    
    printf("==============================\nDevice name: %s\n------------------------------\n", prop.name);
    printf("Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
    printf("Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
    printf("Peak Memory Bandwidth (GB/s): %f\n==============================\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //printf("Max Threads Per Block: %i\n", prop.maxThreadsPerBlock);
    //printf("Max Grid Size: %ix%ix%i\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);

}

//=============================================================================
//                          ALLOCATION
//                    0=UNIFIED ~~ 1=ZERO COPY
//=============================================================================
template<typename Scalar>
void allocation(Scalar **vector_in, Scalar **vector_out, int dim_local, int mem_option)
{
    switch (mem_option)
    {
    case(0):
        hipMallocManaged((void **)vector_in, sizeof(Scalar)*dim_local);
        hipMallocManaged((void **)vector_out, sizeof(Scalar)*dim_local);
        break;
    case(1):
        hipSetDeviceFlags(hipDeviceMapHost);
        hipHostAlloc((void **)vector_in, sizeof(Scalar)*dim_local, hipHostMallocMapped);
        hipHostAlloc((void **)vector_out, sizeof(Scalar)*dim_local, hipHostMallocMapped);
        break;
    }   
}
template void allocation<int>(int **vector_in, int **vector_out, int dim_local, int mem_option);
template void allocation<float>(float **vector_in, float **vector_out, int dim_local, int mem_option);
template void allocation<double>(double **vector_in, double **vector_out, int dim_local, int mem_option);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
//                          KERNEL
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename Scalar>
float gpu_copy_time(Scalar *one, Scalar * two, Scalar *result, int dim_local, int runs, int version, int mem_option)
{
    Timer timer;
    float elapsed_time = 0.0;

    int num_blocks = ceil((double)dim_local / 1024);
    int num_threads = ceil(((double)dim_local / num_blocks) / 32) * 32;

    switch (version)
    {
    case(0) :               //kernel_standart
        if (mem_option == 0)
        {
            //=================================//
            timer.start();
            for (int i = 0; i < runs; i++)
            {
                gpu_copy<<<num_blocks, num_threads>>>(one, two, dim_local);
            }
            hipDeviceSynchronize();
            elapsed_time = timer.stop()*1.0e3;
            //=================================//
        }
        else if (mem_option == 1)
        {
            Scalar *d_one, *d_two;
            hipHostGetDevicePointer((void **)&d_one, (void *)one, 0);
            hipHostGetDevicePointer((void **)&d_two, (void *)two, 0);

            //=================================//
            timer.start();
            for (int i = 0; i < runs; i++)
            {
                gpu_copy << <num_blocks, num_threads, sizeof(double)*num_threads >> >(d_one, d_two, dim_local);
            }
            hipDeviceSynchronize();
            elapsed_time = timer.stop()*1.0e3;
            //=================================//
            
        }
        break;
    }
    return elapsed_time / runs;
}
template float gpu_copy_time<int>(int *one, int * two, int dim_local, int runs, int version, int mem_option);
template float gpu_copy_time<float>(float *one, float * two, int dim_local, int runs, int version, int mem_option);
template float gpu_copy_time<double>(double *one, double * two, int dim_local, int runs, int version, int mem_option);



//GENERATING KERNEL TIME UNIFIED MEMORY
template<typename Scalar>
void gpu_copy_overall(Scalar *one, Scalar * two, int dim_local, int version, int mem_option)
{
    int num_blocks = ceil((double)dim_local / 1024);
    int num_threads = ceil(((double)dim_local / num_blocks) / 32) * 32;

    switch (version)
    {
    case(0) :               //kernel_standart
        if (mem_option == 0)
        {
            gpu_copy << <num_blocks, num_threads >> >(one, two, dim_local);
            hipDeviceSynchronize();
        }
        else if (mem_option == 1)
        {
            hipHostGetDevicePointer((void **)&d_one, (void *)one, 0);
            hipHostGetDevicePointer((void **)&d_two, (void *)two, 0);

            gpu_copy << <num_blocks, num_threads >> >(d_one, d_two, dim_local);
            hipDeviceSynchronize();
        }
    }
}
template void gpu_copy_overall<int>(int *one, int * two, int dim_local, int version, int mem_option);
template void gpu_copy_overall<float>(float *one, float * two, int dim_local, int version, int mem_option);
template void gpu_copy_overall<double>(double *one, double * two, int dim_local, int version, int mem_option);


//=============================================================================
//                              CLEANUP FUNCTIONS
//=============================================================================
template <typename Scalar>
void cleanup(Scalar *one, Scalar *two, int method)
{
    switch(method)
    {
        case(0):
            hipFree(one);
            hipFree(two);
            break;
        case(1):
            hipHostFree(one);
            hipHostFree(two);
            break;
        case(2):
            delete[] one;
            delete[] two;
            break;
    }
}
template void cleanup<int>(int *one, int *two, int method);
template void cleanup<float>(float *one, float *two, int method);
template void cleanup<double>(double *one, double *two, int method);








//====Ich war nicht mutig genug es zu loeschen :D===/
/*

template <typename Scalar>
void cleanupgpu(Scalar *data)
{
hipHostFree(data);
}
template void cleanupgpu<int>(int *data);
template void cleanupgpu<float>(float *data);
template void cleanupgpu<double>(double *data);

//ALLOCATE MEMORY FUNCTION FOR UNIFIED MEMORY for DistEllpack
template<typename Scalar>
void alloc_unifiedD(Scalar **data, int **indices, int dim_local, int dim_local)
{
hipMallocManaged((void **)data, sizeof(Scalar)*dim_local*dim_local);
hipMallocManaged((void **)indices, sizeof(int)*dim_local*dim_local);
}
template void alloc_unifiedD<int>(int **data, int **indices, int dim_local, int dim_local);
template void alloc_unifiedD<float>(float **data, int **indices, int dim_local, int dim_local);
template void alloc_unifiedD<double>(double **data, int **indices, int dim_local, int dim_local);

// ALLOCATE MEMORY FUNCTION FOR UNIFIED MEMORY FOR SLICEDVECTOR
template<typename Scalar>
void alloc_unifiedV(Scalar **fvec, int dim_fvec)
{
hipMallocManaged((void **)fvec, sizeof(Scalar)*dim_fvec);
}
template void alloc_unifiedV<int>(int **fvec, int dim_fvec);
template void alloc_unifiedV<float>(float **fvec, int dim_fvec);
template void alloc_unifiedV<double>(double **fvec, int dim_fvec);
*/
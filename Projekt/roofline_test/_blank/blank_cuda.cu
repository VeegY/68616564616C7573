#include "hip/hip_runtime.h"
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "include/timer.hpp"

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename type>
__global__ void kernel()
{

}

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             ALLOCATION                                  ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename type>
void allocation(type **data, size_t size)
{    
    hipMallocManaged((void **)data, sizeof(type)*size);
}
template void allocation<int>(int **data, size_t size);
template void allocation<float>(float **data, size_t size);
template void allocation<double>(double **data, size_t size);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL CONFIG                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
void generate_config(int *num_threads, int *num_blocks)
{

}

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL TIMING                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
float invoke_gpu_time(int runs)
{
    Timer timer;
    float elapsed_time = 0.0;
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks);

    //=================================//
    timer.start();
    for (int i = 0; i < runs; i++)
    {
        kernel<<<num_blocks, num_threads>>>();
    }
    hipDeviceSynchronize();
    elapsed_time = timer.stop()*1.0e3;
    //=================================//
            
    return elapsed_time / runs;
}
template float invoke_gpu_time<float>(int runs);
template float invoke_gpu_time<double>(int runs);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
void invoke_gpu_overall()
{
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks);

    kernel<<<num_blocks, num_threads>>>();
    hipDeviceSynchronize();
}
template void invoke_gpu_overall<float>();
template void invoke_gpu_overall<double>();



//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             CLEANUP                                     ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template <typename type>
void cleanup(type *data)
{
   hipFree(data);
}
template void cleanup<int>(int *data);
template void cleanup<float>(float *data);
template void cleanup<double>(double *data);

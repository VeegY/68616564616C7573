#include "hip/hip_runtime.h"
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "include/timer.hpp"

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename type>
__global__ void gpu_axpy(type *scalar, type *vector_x, type *vector_y, type *result, int dim)
{
    idx = threadIdx.x + blockIdx.x*blockDim.x;
    type scale = scalar[0];
    if (idx < dim)
    {
        result[idx] = scale*vector_x[idx] + vector_y[idx];
    }

}

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             ALLOCATION                                  ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename type>
void allocation(type **data, size_t size)
{    
    hipMallocManaged((void **)data, sizeof(type)*size);
}
template void allocation<int>(int **data, size_t size);
template void allocation<float>(float **data, size_t size);
template void allocation<double>(double **data, size_t size);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL CONFIG                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
void generate_config(int *num_threads, int *num_blocks)
{

}

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL TIMING                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
float invoke_gpu_time(type scalar, type *vector_x, type *vector_y, type *result, int dim, int runs)
{
    Timer timer;
    float elapsed_time = 0.0;
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks);

    //=================================//
    timer.start();
    for (int i = 0; i < runs; i++)
    {
        gpu_axpy<<<num_blocks, num_threads>>>(scalar, vector_x, vector_y, result, dim);
    }
    hipDeviceSynchronize();
    elapsed_time = timer.stop()*1.0e3;
    //=================================//
            
    return elapsed_time / runs;
}
template float invoke_gpu_time<float>(float scalar, float *vector_x, float *vector_y, float *result, int dim, int runs);
template float invoke_gpu_time<double>(double scalar, double *vector_x, double *vector_y, double *result, int dim, int runs);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
void invoke_gpu_overall(type scalar, type *vector_x, type *vector_y, type *result, int dim)
{
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks);

    gpu_axpy<<<num_blocks, num_threads>>>();
    hipDeviceSynchronize();
}
template void invoke_gpu_overall<float>(float scalar, float *vector_x, float *vector_y, float *result, int dim);
template void invoke_gpu_overall<double>(double scalar, double *vector_x, double *vector_y, double *result, int dim);



//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             CLEANUP                                     ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template <typename type>
void cleanup(type *data)
{
   hipFree(data);
}
template void cleanup<int>(int *data);
template void cleanup<float>(float *data);
template void cleanup<double>(double *data);

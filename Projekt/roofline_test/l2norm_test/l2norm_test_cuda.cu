#include "hip/hip_runtime.h"
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "include/timer.hpp"
#include <cstdio>

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
static __inline__ __device__ double __shfl_down(double var, int laneMask, int width = warpSize)
{
    int hi, lo;
    asm volatile("mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "d"(var));
    hi = __shfl_down(hi, laneMask, width);
    lo = __shfl_down(lo, laneMask, width);
    return __hiloint2double(hi, lo);
}


__inline__ __device__
double warpReduceSum(double val) 
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down(val, offset, warpSize);
    return val;
}


__inline__ __device__
double blockReduceSum(double val) 
{

    static __shared__ double shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane == 0) shared[wid] = val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

                                  //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

    if (wid == 0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}


__global__ void deviceReduceKernel(double *in, double *out, int N) 
{
  double sum = 0;
  double value = 0;
  //reduce multiple elements per thread
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
       i < N; 
       i += blockDim.x * gridDim.x) 
  {
      value = in[i];
      sum += value*value;
  }
  sum = blockReduceSum(sum);
  if (threadIdx.x==0)
    out[blockIdx.x]=sum;
}


/*template<typename type>
__global__ void kernel(type *vector, type *placehold, int dim_local)
{
    extern __shared__ double array[];
    type* shar = (type*)array;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int sidx = threadIdx.x;
    type value = (type)0;
    if (idx < dim_local)
    {
        value = vector[idx];
        value *= value;
    }
    shar[sidx] = value;
    __syncthreads();

    //reduce kernel
    for (int offset = blockDim.x / 2; offset >0; offset >>= 1)
    {
        if (sidx < offset)
        {
            shar[sidx] += shar[sidx + offset];
        }
        __syncthreads();
    }

    if (sidx == 0)
    {
        placehold[blockIdx.x] = shar[0];
    }
}

template<typename type>
__global__ void resultreduce(type *result, type *placehold, int num_blocks)
{
    type value = (type)0;
    for (int i = 0; i < num_blocks; i++)
    {
        value += placehold[i];
    }
    result[0] = sqrt(value);
}*/

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             ALLOCATION                                  ///
///////////////////////////////////////////////////////////////////////////////
//=============================================================================
template<typename type>
void allocation(type **data, size_t size)
{    
    hipMallocManaged((void **)data, sizeof(type)*size);
}
template void allocation<int>(int **data, size_t size);
template void allocation<float>(float **data, size_t size);
template void allocation<double>(double **data, size_t size);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL CONFIG                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
void generate_config(int *num_threads, int *num_blocks, int dim)
{
    
    *num_threads = 1024;
    if (dim<1024)
    {
        int n = dim - 1;
        n |= n >> 1;
        n |= n >> 2;
        n |= n >> 4;
        n |= n >> 8;
        n |= n >> 16;
        n |= n >> 16;
        n |= n >> 16;
        *num_threads = n + 1;
    }
    *num_blocks = ceil((double)dim / 1024);
}

//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL TIMING                               ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
float invoke_gpu_time(type *vector, type *result, int dim, int runs)
{
    Timer timer;
    float elapsed_time = 0.0;
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks, dim);

    type *placehold = NULL;
    hipMallocManaged((void **)&placehold, sizeof(type)*num_blocks);

    //=================================//
    timer.start();
    for (int i = 0; i < runs; i++)
    {
        deviceReduceKernel << <num_blocks, num_threads >> >(vector, result, dim);
        //kernel<<<num_blocks, num_threads, sizeof(double)*num_threads>>>(vector,placehold,dim);
        //resultreduce<<<1, 1>>>(result, placehold, num_blocks);
    }
    hipDeviceSynchronize();
    elapsed_time = timer.stop()*1.0e3;
    //=================================//
            
    return elapsed_time / runs;
}
//template float invoke_gpu_time<float>(float *vector, float *result, int dim, int runs);
template float invoke_gpu_time<double>(double *vector, double *result, int dim, int runs);


//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             KERNEL                                      ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template<typename type>
void invoke_gpu_overall(type *vector, type *result, int dim)
{
    int num_threads, num_blocks;
    generate_config(&num_threads, &num_blocks, dim);

    type *placehold = NULL;
    hipMallocManaged((void **)&placehold, sizeof(type)*num_blocks);

    //kernel<<<num_blocks, num_threads, sizeof(double)*num_threads >>>(vector, placehold, dim);
    //resultreduce << <1, 1 >> >(result, placehold, num_blocks);
    
    hipDeviceSynchronize();
}
template void invoke_gpu_overall<float>(float *vector, float *result, int dim);
template void invoke_gpu_overall<double>(double *vector, double *result, int dim);



//=============================================================================
///////////////////////////////////////////////////////////////////////////////
///                             CLEANUP                                     ///
///////////////////////////////////////////////////////////////////////////////                       
//=============================================================================
template <typename type>
void cleanup(type *data)
{
   hipFree(data);
}
template void cleanup<int>(int *data);
template void cleanup<float>(float *data);
template void cleanup<double>(double *data);

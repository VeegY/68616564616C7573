#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void fib_gpu(int *fib)
{
    int temp=fib[1];
    fib[1]+=fib[0];
    fib[0]=temp;
}

extern  void Fibonacci(int* h_fib)
{


     
    
     int *d_fib;

     if(hipSuccess != hipMalloc(&d_fib,sizeof(int)*2))
     {
          cout << "allocate error" << endl;
     }
     
     if(hipSuccess != hipMemcpy(d_fib, h_fib, sizeof(int)*2, hipMemcpyHostToDevice))
     {
          cout << "copy error" << endl;
     }

     fib_gpu<<<1, 1>>>(d_fib);
        
     if(hipSuccess != hipGetLastError())
     {
          cout << "kernel launch failed" << endl;
     }
     
     
     if(hipSuccess != hipMemcpy(h_fib, d_fib, sizeof(int)*2, hipMemcpyDeviceToHost))
     {
          cout << "copy error" << endl;
     }     
//=====================Kernel=============================//

  //GPUtimer gtimer;
  //gtimer.start();

  //double gtime = gtimer.stop();
  //time[1] = gtime;

  hipDeviceSynchronize();
}

#include "hip/hip_runtime.h"
#include <seqan/parallel.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include "timer.hpp"
#define N 5


template<typename Scalar>
__global__ void  gpu_ax(Scalar *data, Scalar* fvec, Scalar* result, int *indices)
{
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    int row = blockIdx.x;

    Scalar value = 0;
    if(!data[idx] == 0)
    {
      value = data[idx]*fvec[indices[idx]];
      atomicAdd(&result[row],value);
    }

}


__global__ void  gpu_ax(double *data, double* fvec, double* result, int *indices)
{
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    int row = blockIdx.x;

    double value = 0;
    if(!data[idx] == 0)
    {
      value = data[idx]*fvec[indices[idx]];
      atomicCAS(&result[row],result[row],result[row]+value);
    }
}

template<typename Scalar>
void mult_vec_unified(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local)
{
//    Timer timer;
//    timer.start();

    hipMallocManaged(&data, sizeof(Scalar)*max_row_length*dim_local);
    hipMallocManaged(&fvec, sizeof(Scalar)*N);
    hipMallocManaged(&result, sizeof(Scalar)*dim_local);
    hipMallocManaged(&indices, sizeof(int)*max_row_length*dim_local);

    gpu_ax<<<dim_local,max_row_length>>>(data,fvec,result,indices);
    cudaDeviceSyncrhonize();

//    float elapsed = timer.stop();
//    printf("unified memory takes %f ms to complete with max row length %i and dim local %i \n", elapsed,max_row_length,dim_local);
}
template void mult_vec_unified<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local);
template void mult_vec_unified<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local);
template void mult_vec_unified<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local);


template<typename Scalar>
void mult_vec_zero(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local)
{
    Scalar *data_map, *fvec_map, *result_map;
    int *indices_map;

    hipGetDeviceProperties(&prop,0);
    if(prop.canMapHostMemory)
    {
      hipSetDeviceFlags(hipDeviceMapHost);

      hipHostAlloc(&data, sizeof(Scalar)*max_row_length*dim_local, hipHostMallocMapped);
      hipHostAlloc(&fvec, sizeof(Scalar)*N, hipHostMallocMapped);
      hipHostAlloc(&result, sizeof(Scalar)*dim_local, hipHostMallocMapped);
      hipHostAlloc(&indices, sizeof(int)*max_row_length*dim_local, hipHostMallocMapped);

      hipHostGetDevicePointer(&data_map, data, 0);
      hipHostGetDevicePointer(&fvec_map, fvec, 0);
      hipHostGetDevicePointer(&result_map, result, 0);
      hipHostGetDevicePointer(&indices_map, indices, 0);

      gpu_ax<<<dim_local,max_row_length>>>(data_map,fvec_map,result_map,indices_map);

//      float elapsed = timer.stop();
//      printf("zero copy takes %f ms to complete with max row length %i and dim local %i \n", elapsed,max_row_length,dim_local);
    }
}
template void mult_vec_zero<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local);
template void mult_vec_zero<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local);
template void mult_vec_zero<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local);


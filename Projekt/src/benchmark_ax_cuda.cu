#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "include/timer.hpp"
#define RESET "\e[0m"
#define BLUE "\e[34;1m"
#define CYAN "\e[36;1m"
#define GREY "\e[30;1m"
#define MAGENTA "\e[35;1m"


template <typename Scalar>
void cleanup(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int method);

//KERNEL
template<typename type>
__global__ void  gpu_ax(type* data, type* fvec, type* result, int* indices, int max_row_length, int dim_local)
{

    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if(idx<dim_local)
    {
      int col;
      type svalue = 0, value;
      for(int i = 0;i < max_row_length; i++)
      {
        value = data[i*dim_local+idx];
        col = indices[i*dim_local+idx];
        svalue += value*fvec[col];
      }
      result[idx]=svalue;
    }
}


//CALCULATING MEMORY BANDWITH
template<typename type>
void performance(int max_row_length, int dim_local, float time_ku, float time_ou, float time_kz, float time_oz, int runs, type schalter)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);

    //===#ELEMENTS IN THE MATRIX===================================//
    unsigned long long int elements = 7 * dim_local - 2 - 2 * (floor(pow(dim_local, (1.0 / 3.0)))) - 2 * (floor(pow(dim_local, (2.0 / 3.0))));

    //==='DISK STORAGE~============================================//
    unsigned long long int storage = sizeof(type)*(2 * dim_local + dim_local*max_row_length) + sizeof(int)*dim_local*max_row_length;
    
        //===#FLOP=====================================================//
    unsigned long long int flop = 2 * elements;

    //==#BYTES=====================================================//           
    int bytes = elements*(sizeof(type) + sizeof(int)) + 2*(sizeof(type)*dim_local);// Elements(Data+Indices) + Fvec Read und Result Write

    printf(GREY "===============================================\n");
    printf(MAGENTA "                PERFORMANCE\n");
    printf("        DIM = %i ~~ %i Iterations\n", dim_local, runs);
    printf("            %.2fGB/2GB DRAM used\n", storage / 1.0e9);
    printf(GREY "===============================================\n");
    printf("-----------------------------------------------\n");
    printf(CYAN "                UNIFIED_MERMORY\n");
    printf(GREY "-----------------------------------------------\n");
    printf(CYAN "Kernel Runtime:\t\t\t%f(ms)\n",time_ku);
    printf("Overall Runtime:\t\t%f(ms)\n",time_ou*1.0e3);
    printf("Bandwith(th. Peak):\t\t%.2f(14.9)(GB/s)\n", bytes / (time_ku*1.0e6));
    printf("Flops(th. Peak):\t\t%.2f(326)(GFLOPS/s)\n", flop  / (time_ku*1.0e6));
    printf(GREY "-----------------------------------------------\n");
    printf("-----------------------------------------------\n");
    printf(BLUE "                ZERO_COPY\n");
    printf(GREY "-----------------------------------------------\n");
    printf(BLUE "Kernel Runtime:\t\t\t%f(ms)\n",time_kz);
    printf("Overall Runtime:\t\t%f(ms)\n",time_oz*1.0e3);
    printf("Bandwith(th. Peak):\t\t%.2f(14.9)(GB/s)\n", bytes / (time_kz*1.0e6));
    printf("Flops(th. Peak):\t\t%.2f(326)(GFLOPS/s)\n", flop  / (time_kz*1.0e6));
    printf(GREY "-----------------------------------------------\n" RESET);



    
}
template void performance<int>(int max_row_length, int dim_local, float time_ku, float time_ou, float time_kz, float time_oz, int runs, int schalter);
template void performance<float>(int max_row_length, int dim_local, float time_ku, float time_ou, float time_kz, float time_oz, int runs, float schalter);
template void performance<double>(int max_row_length, int dim_local, float time_ku, float time_ou, float time_kz, float time_oz, int runs, double schalter);


//PROPERTIES OF TEGRA K1
void print_p()
{

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    
    printf("==============================\nDevice name: %s\n------------------------------\n", prop.name);
    printf("Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
    printf("Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
    printf("Peak Memory Bandwidth (GB/s): %f\n==============================\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //printf("Max Threads Per Block: %i\n", prop.maxThreadsPerBlock);
    //printf("Max Grid Size: %ix%ix%i\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);

}

//ALLOCATE MEMORY FUNCTION FOR UNIFIED MEMORY
template<typename Scalar>
void alloc_unified(Scalar **data, Scalar **fvec, Scalar **result, int **indices, int max_row_length, int dim_local,int dim_fvec)
{
    hipMallocManaged((void **)data, sizeof(Scalar)*dim_local*max_row_length);
    hipMallocManaged((void **)fvec, sizeof(Scalar)*dim_fvec);
    hipMallocManaged((void **)result, sizeof(Scalar)*dim_local);
    hipMallocManaged((void **)indices, sizeof(int)*dim_local*max_row_length);
}
template void alloc_unified<int>(int **data, int **fvec, int **result, int **indices, int max_row_length, int dim_local, int dim_fvec);
template void alloc_unified<float>(float **data, float **fvec, float **result, int **indices, int max_row_length, int dim_local, int dim_fvec);
template void alloc_unified<double>(double **data, double **fvec, double **result, int **indices, int max_row_length, int dim_local, int dim_fvec);


//ALLOCATE MEMORY FUNCTION FOR UNIFIED MEMORY for DistEllpack
template<typename Scalar>
void alloc_unifiedD(Scalar **data, int **indices, int max_row_length, int dim_local)
{
    hipMallocManaged((void **)data, sizeof(Scalar)*dim_local*max_row_length);
    hipMallocManaged((void **)indices, sizeof(int)*dim_local*max_row_length);
}
template void alloc_unifiedD<int>(int **data, int **indices, int max_row_length, int dim_local);
template void alloc_unifiedD<float>(float **data, int **indices, int max_row_length, int dim_local);
template void alloc_unifiedD<double>(double **data, int **indices, int max_row_length, int dim_local);

// ALLOCATE MEMORY FUNCTION FOR UNIFIED MEMORY FOR SLICEDVECTOR
template<typename Scalar>
void alloc_unifiedV(Scalar **fvec, int dim_fvec)
{
    hipMallocManaged((void **)fvec, sizeof(Scalar)*dim_fvec);
}
template void alloc_unifiedV<int>(int **fvec, int dim_fvec);
template void alloc_unifiedV<float>(float **fvec, int dim_fvec);
template void alloc_unifiedV<double>(double **fvec, int dim_fvec);


//ALLOCATE MEMORY FUNCTION FOR ZERO COPY 
template<typename Scalar>
void alloc_zero(Scalar **data, Scalar **fvec, Scalar **result, int ** indices, int max_row_length, int dim_local, int dim_fvec)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);

    if(prop.canMapHostMemory)
    {
      hipSetDeviceFlags(hipDeviceMapHost);

      hipHostAlloc((void **)data, sizeof(Scalar)*max_row_length*dim_local, hipHostMallocMapped);
      hipHostAlloc((void **)fvec, sizeof(Scalar)*dim_fvec, hipHostMallocMapped);
      hipHostAlloc((void **)result, sizeof(Scalar)*dim_local, hipHostMallocMapped);
      hipHostAlloc((void **)indices, sizeof(int)*max_row_length*dim_local, hipHostMallocMapped);
    }
}
template void alloc_zero<int>(int **data, int **fvec, int **result, int **indices, int max_row_length, int dim_local, int dim_fvec);
template void alloc_zero<float>(float **data, float **fvec, float **result, int **indices, int max_row_length, int dim_local, int dim_fvec);
template void alloc_zero<double>(double **data, double **fvec, double **result, int **indices, int max_row_length, int dim_local, int dim_fvec);

//=============================================================================
//                          UNIFIED KERNEL FUNCTIONS
//=============================================================================

//GENERATING KERNEL TIME UNIFIED MEMORY
template<typename Scalar>
float mult_vec_unified_time(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local, int dim_fvec, int runs)
{
    hipEvent_t start_unified, stop_unified;
    hipEventCreate(&start_unified);
    hipEventCreate(&stop_unified);
    
    int num_blocks = ceil((double)dim_local/1024);
    int num_threads = ceil(((double)dim_local/num_blocks)/32)*32;
    
    hipEventRecord(start_unified);
    for (int i = 0; i < runs; i++)
    {
        gpu_ax<<<num_blocks,num_threads>>>(data,fvec,result,indices,max_row_length, dim_local);
        
    }
    hipEventRecord(stop_unified);
    
    hipEventSynchronize(stop_unified);
    float elapsedTime_unified = 0.0;
    hipEventElapsedTime(&elapsedTime_unified, start_unified, stop_unified);

    hipEventDestroy(start_unified);
    hipEventDestroy(stop_unified);
    return (elapsedTime_unified / (float)runs);
}
template float mult_vec_unified_time<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local,int dim_fvec, int runs);
template float mult_vec_unified_time<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local, int dim_fvec, int runs);
template float mult_vec_unified_time<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local, int dim_fvec, int runs);


//GENERATING KERNEL TIME UNIFIED MEMORY
template<typename Scalar>
void mult_vec_unified(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local, int dim_fvec)
{
    int num_blocks = ceil((double)dim_local/1024);
    int num_threads = ceil(((double)dim_local/num_blocks)/32)*32;

    gpu_ax<<<num_blocks,num_threads>>>(data,fvec,result,indices,max_row_length, dim_local);
    hipDeviceSynchronize();
}
template void mult_vec_unified<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local,int dim_fvec);
template void mult_vec_unified<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local, int dim_fvec);
template void mult_vec_unified<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local, int dim_fvec);


//=============================================================================
//                              ZERO KERNEL FUNCTIONS
//=============================================================================

//KERNE CALL WITH ZERO COPY (NEED TO CALL ALLOC_ZERO BEFORE)
template<typename Scalar>
float mult_vec_zero_time(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local, int dim_fvec, int runs)
{
    hipEvent_t start_zero, stop_zero;
    hipEventCreate(&start_zero);
    hipEventCreate(&stop_zero);

    Scalar *d_data, *d_fvec, *d_result;
    int *d_indices;

    hipHostGetDevicePointer((void **)&d_data,(void *)data, 0);
    hipHostGetDevicePointer((void **)&d_fvec, (void *)fvec, 0);
    hipHostGetDevicePointer((void **)&d_result, (void *)result, 0);
    hipHostGetDevicePointer((void **)&d_indices, (void *)indices, 0);

    int num_blocks = ceil((double)dim_local/1024);
    int num_threads = ceil(((double)dim_local/num_blocks)/32)*32;

    hipEventRecord(start_zero);
    for (int i=0;i<runs;i++)
    {
        gpu_ax<<<num_blocks,num_threads>>>(d_data, d_fvec, d_result, d_indices, max_row_length, dim_local);
   
    }
    hipEventRecord(stop_zero);

    hipEventSynchronize(stop_zero);
    float elapsedTime_zero = 0.0;
    hipEventElapsedTime(&elapsedTime_zero, start_zero, stop_zero);

    hipEventDestroy(start_zero);
    hipEventDestroy(stop_zero);
    cleanup(d_data, d_fvec, d_result, d_indices, 0);
    return (elapsedTime_zero /(float)runs);
}
template float mult_vec_zero_time<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local, int  dim_fvec, int runs);
template float mult_vec_zero_time<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local, int dim_fvec, int runs);
template float mult_vec_zero_time<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local, int dim_fvec, int runs);


template<typename Scalar>
void mult_vec_zero(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int max_row_length, int dim_local, int dim_fvec)
{
    Scalar *d_data, *d_fvec, *d_result;
    int *d_indices;

    hipHostGetDevicePointer((void **)&d_data,(void *)data, 0);
    hipHostGetDevicePointer((void **)&d_fvec, (void *)fvec, 0);
    hipHostGetDevicePointer((void **)&d_result, (void *)result, 0);
    hipHostGetDevicePointer((void **)&d_indices, (void *)indices, 0);

    int num_blocks = ceil((double)dim_local/1024);
    int num_threads = ceil(((double)dim_local/num_blocks)/32)*32;

    gpu_ax<<<num_blocks,num_threads>>>(d_data, d_fvec, d_result, d_indices, max_row_length, dim_local);
    hipDeviceSynchronize();
    cleanup(d_data, d_fvec, d_result, d_indices, 0);
}
template void mult_vec_zero<int>(int* data, int* fvec, int* result, int* indices, int max_row_length, int dim_local, int  dim_fvec);
template void mult_vec_zero<float>(float* data, float* fvec, float* result, int* indices, int max_row_length, int dim_local, int dim_fvec);
template void mult_vec_zero<double>(double* data, double* fvec, double* restult, int* indices, int max_row_length, int dim_local, int dim_fvec);


//=============================================================================
//                              CLEANUP FUNCTIONS
//=============================================================================
template <typename Scalar>
void cleanup(Scalar *data, Scalar *fvec, Scalar *result, int *indices, int method)
{
    switch(method)
    {
        case(0):
            hipFree(data);
            hipFree(fvec);
            hipFree(result);
            hipFree(indices);
            break;
        case(1):
            hipHostFree(data);
            hipHostFree(fvec);
            hipHostFree(result);
            hipHostFree(indices);
            break;
        case(2):
            delete[] data;
            delete[] fvec;
            delete[] result;
            delete[] indices;
            break;
    }
}
template void cleanup<int>(int *data, int *fvec, int *result, int *indices, int method);
template void cleanup<float>(float *data, float *fvec, float *result, int *indices, int method);
template void cleanup<double>(double *data, double *fvec, double *result, int *indices, int method);


template <typename Scalar>
void cleanupgpu(Scalar *data)
{   
    hipHostFree(data);
}
template void cleanupgpu<int>(int *data);
template void cleanupgpu<float>(float *data);
template void cleanupgpu<double>(double *data);
